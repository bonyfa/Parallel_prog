#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <chrono>
#include <stdexcept>
#include <iomanip>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

// CUDA kernel для умножения матриц
__global__ void matrixMultiplyKernel(const double* A, const double* B, double* C, 
                                    int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double sum = 0.0;
        for (int k = 0; k < colsA; ++k) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

class Matrix {
private:
    vector<vector<double>> data;
    int rows;
    int cols;

public:
    Matrix() : rows(0), cols(0) {}
    
    Matrix(int r, int c) : rows(r), cols(c), data(r, vector<double>(c)) {}
    
    void readFromFile(const string& filename) {
        ifstream file(filename);
        if (!file.is_open()) {
            throw runtime_error("Не удалось открыть файл: " + filename);
        }

        string line;
        while (getline(file, line)) {
            if (line.empty()) continue;

            vector<double> row;
            stringstream ss(line);
            double val;
            while (ss >> val) {
                row.push_back(val);
            }

            if (cols == 0) {
                cols = row.size();
            } else if (row.size() != cols) {
                throw runtime_error("Несогласованное количество столбцов в файле " + filename);
            }
            data.push_back(row);
        }
        rows = data.size();
    }

    void writeToFile(const string& filename, long long duration_ms = -1) const {
        ofstream file(filename);
        if (!file.is_open()) {
            throw runtime_error("Не удалось открыть файл: " + filename);
        }

        if (duration_ms >= 0) {
            file << "Время выполнения: " << duration_ms << " мс\n";
            file << "Размер матрицы: " << rows << "x" << cols << "\n\n";
        }

        // Выводим только первые 10x10 элементов для больших матриц
        int outputRows = min(10, rows);
        int outputCols = min(10, cols);

        for (int i = 0; i < outputRows; i++) {
            for (int j = 0; j < outputCols; j++) {
                file << fixed << setprecision(6) << data[i][j] << " ";
            }
            file << "\n";
        }
    }

    Matrix multiply(const Matrix& other) const {
        if (cols != other.rows) {
            throw runtime_error("Несовместимые размеры матриц для умножения");
        }

        Matrix result(rows, other.cols);

        // Подготовка данных для CUDA
        double *d_A, *d_B, *d_C;
        size_t sizeA = rows * cols * sizeof(double);
        size_t sizeB = other.rows * other.cols * sizeof(double);
        size_t sizeC = rows * other.cols * sizeof(double);

        // Выделение памяти на GPU
        hipMalloc(&d_A, sizeA);
        hipMalloc(&d_B, sizeB);
        hipMalloc(&d_C, sizeC);

        // Преобразование данных в одномерные массивы
        vector<double> flatA(rows * cols);
        vector<double> flatB(other.rows * other.cols);
        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < cols; ++j) {
                flatA[i * cols + j] = data[i][j];
            }
        }
        for (int i = 0; i < other.rows; ++i) {
            for (int j = 0; j < other.cols; ++j) {
                flatB[i * other.cols + j] = other.data[i][j];
            }
        }

        // Копирование данных на GPU
        hipMemcpy(d_A, flatA.data(), sizeA, hipMemcpyHostToDevice);
        hipMemcpy(d_B, flatB.data(), sizeB, hipMemcpyHostToDevice);

        // Настройка размеров блоков и сетки
        dim3 blockDim(16, 16);
        dim3 gridDim((other.cols + blockDim.x - 1) / blockDim.x,
                    (rows + blockDim.y - 1) / blockDim.y);

        // Запуск CUDA kernel
        matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, rows, cols, other.cols);

        // Копирование результата обратно на CPU
        vector<double> flatC(rows * other.cols);
        hipMemcpy(flatC.data(), d_C, sizeC, hipMemcpyDeviceToHost);

        // Преобразование результата обратно в двумерный массив
        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < other.cols; ++j) {
                result.data[i][j] = flatC[i * other.cols + j];
            }
        }

        // Освобождение памяти GPU
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        return result;
    }

    int getRows() const { return rows; }
    int getCols() const { return cols; }
};

int main() {
    try {
        Matrix A, B;
        
        cout << "Чтение матрицы A..." << endl;
        A.readFromFile("matrix_a.txt");
        cout << "Матрица A: " << A.getRows() << "x" << A.getCols() << endl;

        cout << "Чтение матрицы B..." << endl;
        B.readFromFile("matrix_b.txt");
        cout << "Матрица B: " << B.getRows() << "x" << B.getCols() << endl;

        if (A.getCols() != B.getRows()) {
            throw runtime_error("Размеры матриц не подходят для умножения");
        }

        cout << "Умножение матриц с использованием CUDA..." << endl;
        auto start = high_resolution_clock::now();
        Matrix C = A.multiply(B);
        auto end = high_resolution_clock::now();

        auto duration = duration_cast<milliseconds>(end - start);
        cout << "Время умножения: " << duration.count() << " мс" << endl;

        cout << "Запись результата в result.txt..." << endl;
        C.writeToFile("result.txt", duration.count());
        cout << "Готово!" << endl;

    } catch (const exception& e) {
        cerr << "Ошибка: " << e.what() << endl;
        return 1;
    }

    return 0;
} 